#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
//#include "cutil.h"

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        printf("Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        printf("\nPress ENTER to exit...\n");
        getchar();
        exit(-1);
    }
}
//name of the input file
#define INPUT_FILE_NAME "input.txt"
//name of the compressed file
#define COMPRESSED_FILE_NAME "compressed.txt"
#define COMPRESSED_FILE_NAME_GPU "compressed_gpu.txt"

//name of the uncompressed file
#define DECOMPRESSED_FILE_NAME "decompressed.txt"
//name of the config file
#define  CONFIG_FILE_NAME "config.txt"
//max number of characters
#define MAX_CHAR 256

//#define MAX_CHAR 30
//max lenght of the number which can occur in char_frequency or char_huffman_table)
#define MAX_LENGTH_OF_NUMBER 10
//lenght of the array in shared memory on device
#define SHARED_MEMORY_SIZE 256
//lenght of the array in const memory on device
#define CONST_MEMORY_SIZE 15000 //(MAX_CHAR*(MAX_CHAR-1))

//To fill and pass the file as an array to GPU
#define MAX_FILE_CHARS 50000
#define BLOCK_SIZE 256


struct node {
   int val;
   int weight;
   struct node * right, * left;
};

//keeps frequency of particular characters (index - symbof of the character, value - frequency of the character)
int char_frequency[MAX_CHAR];
//keeps huffman table
int char_huffman_table[MAX_CHAR][MAX_CHAR-1];
//keeps number which tells how many bits were unused in last byte (variable is set after call compress_file())
int last_byte_padding=0;

//for writing gpu output
int last_byte_padding_gpu = 0;


//keeps number of characters in current input file - file has to have less than 2,147,483,647 characters (variable is set after call read_file())
int number_of_char=0 ;

//To fill and pass the file as an array to GPU
unsigned char *h_input=0,*d_input=0;

// To read char_huffman_table at the GPU
int *d_char_huffman_table=0;

int copiedarray2[MAX_CHAR][MAX_CHAR-1];


__device__ int char_huffman_table_gpu[MAX_CHAR][MAX_CHAR-1];

//To write the output from compression in GPU

 //char *compressedfile_array=0;

 bool *compressedfile_array=0;

 bool *finalcompressed_array=0;

 // To keep track of how many characters each block wrote

 int *block_cntr_array=0;
 int *block_cntr_array_check=0;
 int *d_last_byte_padding=0;

 int *finalsize=0;
 int *orig_number_of_char=0;
 int *huffman_check = (int *)malloc((MAX_CHAR)*(MAX_CHAR-1) *sizeof(int));

 bool *d_bool = 0;

 bool *h_bool = 0;



 __global__ void final_compression(int *block_cntr_array,bool *compressedfile_array,bool *finalcompressed_array,int number_of_char)
 //__device__ void final_compression(int *block_cntr_array,bool *compressedfile_array,bool *finalcompressed_array)
{
int index_blocks=blockIdx.x*blockDim.x+threadIdx.x;
int index_file=(blockIdx.x*blockDim.x+threadIdx.x)*255;
int final_index=0;

if(index_blocks < number_of_char)
{
for(int i=0;i<index_blocks;i++)
{
final_index = final_index+ block_cntr_array[i];
}
for(int i=0;i<block_cntr_array[index_blocks];i++)
{
finalcompressed_array[final_index+i]=compressedfile_array[index_file+i];
}

}
}


//__global__ void computearray_size(int* block_cntr_array,int *finalsize,int *orig_number_of_char)
__device__ void computearray_size(int* block_cntr_array,int *finalsize,int *orig_number_of_char)
{
*finalsize = 0;
for(int i=0;i<*orig_number_of_char;i++)
{
(*finalsize)=(*finalsize) + block_cntr_array[i];
}

}



/*__global__ void compress_file_gpu(unsigned char *d_input,char *compressedfile_array,int *char_huffman_table2,int *block_cntr_array,int* d_last_byte_padding)
{
	int write_counter=0,block_counter=0;	//how many bits have been written in specific byte
	 unsigned char input_char;
	unsigned char output_char = 0x0;
	 unsigned char end_of_file = 255;
	unsigned char mask = 0x01; //00000001;
	int index_file=(blockIdx.x*blockDim.x+threadIdx.x)*255;
	int index_blocks=blockIdx.x*blockDim.x+threadIdx.x;
	

	//for(int i=0;i<MAX_CHAR;i++)
	//{
		//int *row = (int*)((char*)char_huffman_table2 + i * pitch);
		//for (int c = 0; c < MAX_CHAR-1; ++c) {
          //   char_huffman_table_gpu[i][c] = row[c];
        //}
	//}

  			input_char = d_input[index_blocks];			
			for(int i = 0 ; i < (MAX_CHAR - 1) ; i++)				
			{
				if(char_huffman_table2[input_char*255+i] == 0)			//detect if current character on particular position has 0 or 1
				{
					output_char = output_char << 1;					//if 0 then shift bits one position to left (last bit after shifting is 0)
					write_counter++;
					block_counter++;
				}
				else if(char_huffman_table2[input_char*255+i] == 1)
				{
					output_char = output_char << 1;					//if 1 then shift bits one position to left...
					output_char = output_char | mask;				//...and last bit change to: 1
					write_counter++;
					block_counter++;
				}
				else //-1
				{
					//if(input_char == end_of_file)					//if EOF is detected then write current result to file
					//{													
						if(write_counter != 0)
						{
							output_char = output_char << (8-write_counter);
							compressedfile_array[index_file]=output_char;
							output_char = 0x0;
						}
						else	//write_counter == 0
						{
							compressedfile_array[index_file]=output_char;
						}
					//}

					break;
				}

				if(write_counter == 8)								//if result achieved 8 (size of char) then write it to compressed_file
				{
					compressedfile_array[index_file]=output_char;
					output_char = 0x0;
					write_counter = 0;
				}
			}
		
		block_cntr_array[index_blocks]=block_counter;
		*d_last_byte_padding = write_counter;							//to decompress file we have to know how many bits in last byte have been written
		//update_config(write_counter); //TODO to zakomentowac przy ostatecznych pomiarach
	
}*/

//__global__ void compress_file_gpu(unsigned char *d_input,bool *compressedfile_array,int *char_huffman_table2,int *block_cntr_array,int* d_last_byte_padding)
__global__ void compress_file_gpu(unsigned char *d_input,bool *compressedfile_array,int *char_huffman_table2,int *block_cntr_array,int* d_last_byte_padding,int *finalsize,int *orig_number_of_char,int number_of_char)
{
	//int write_counter=0,
	int block_counter=0;	//how many bits have been written in specific byte
	 unsigned char input_char;
	//unsigned char output_char = 0x0;
	 //unsigned char end_of_file = 255;
	//unsigned char mask = 0x01; //00000001;
	int index_file=(blockIdx.x*blockDim.x+threadIdx.x)*255;
	int index_blocks=blockIdx.x*blockDim.x+threadIdx.x;
	
	if(index_blocks < number_of_char) 
	{
	//for(int i=0;i<MAX_CHAR;i++)
	//{
		//int *row = (int*)((char*)char_huffman_table2 + i * pitch);
		//for (int c = 0; c < MAX_CHAR-1; ++c) {
          //   char_huffman_table_gpu[i][c] = row[c];
        //}
	//}

  			input_char = d_input[index_blocks];			
			for(int i = 0 ; i < (MAX_CHAR - 1) ; i++)				
			{
				if(char_huffman_table2[input_char*255+i] == 0)			//detect if current character on particular position has 0 or 1
				{
					//output_char = output_char << 1;					//if 0 then shift bits one position to left (last bit after shifting is 0)
					compressedfile_array[index_file+i] = false;
					//write_counter++;
					block_counter++;
				}
				else if(char_huffman_table2[input_char*255+i] == 1)
				{
					//output_char = output_char << 1;					//if 1 then shift bits one position to left...
					//output_char = output_char | mask;				//...and last bit change to: 1
					//write_counter++;
					compressedfile_array[index_file+i] = true;
					block_counter++;
				}
				else //-1
				{
					/*if(input_char == end_of_file)					//if EOF is detected then write current result to file
					{													
						if(write_counter != 0)
						{
							output_char = output_char << (8-write_counter);
							compressedfile_array[index_file]=output_char;
							output_char = 0x0;
						}
						else	//write_counter == 0
						{
							compressedfile_array[index_file]=output_char;
						}
					}*/

					break;
				}

				/*if(write_counter == 8)								//if result achieved 8 (size of char) then write it to compressed_file
				{
					compressedfile_array[index_file]=output_char;
					output_char = 0x0;
					write_counter = 0;
				}*/
			}
		
		block_cntr_array[index_blocks]=block_counter;
		//*d_last_byte_padding = write_counter;							//to decompress file we have to know how many bits in last byte have been written
		//update_config(write_counter); //TODO to zakomentowac przy ostatecznych pomiarach

		computearray_size(block_cntr_array,finalsize,orig_number_of_char);
		//final_compression(block_cntr_array,compressedfile_array,finalcompressed_array);

	}
	
}


void write_GPU_compressed(bool *final_compressed_cpu,int *finalsize_cpu)
{
FILE *compressed_file;
int write_counter=0;	//how many bits have been written in specific byte
//unsigned char input_char;
unsigned char output_char = 0x0;
//unsigned char end_of_file = 255;
unsigned char mask = 0x01; //00000001;

compressed_file = fopen(COMPRESSED_FILE_NAME_GPU, "wb");	

if ((compressed_file==NULL))
	{
		perror ("Error reading file");		
	}
else
	{
	for(int i = 0 ; i <  (*finalsize_cpu) ; i++)				
			{
			
			if(int(final_compressed_cpu[i]) == 0)
			{
				output_char = output_char << 1;					//if 0 then shift bits one position to left (last bit after shifting is 0)
					write_counter++;
			}
			else if (int(final_compressed_cpu[i]) == 1)
				{
					output_char = output_char << 1;					//if 1 then shift bits one position to left...
					output_char = output_char | mask;				//...and last bit change to: 1
					write_counter++;
				}
			if(write_counter == 8)								//if result achieved 8 (size of char) then write it to compressed_file
				{
					printf("Compressed char in decimal is %d \n", output_char);
					putc(output_char, compressed_file);				
					output_char = 0x0;
					write_counter = 0;
				}	
				
			}
		
		if(write_counter != 0)
						{
							output_char = output_char << (8-write_counter);
							printf("Compressed char in decimal is %d \n", output_char);
							putc(output_char, compressed_file);
							output_char = 0x0;
						}
	
	}
	fclose(compressed_file);
	last_byte_padding_gpu = write_counter;	
}

























void print_dchar_huffman_table()
{

		printf("\n dchar huffman table ");
		getchar();
		bool flag = false;

	printf("+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++\n");
	printf("Huffman table:\n");
	printf("+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++\n");
	
	for(int i = 0 ; i < MAX_CHAR ; i++)
	{		
		flag = false;
		for(int j = 0 ; j < (MAX_CHAR -1) ; j++)
		{		
			if(copiedarray2[i][j] != -1)
			{				
				if(!flag)
				{
					if(i == 10)//new line
					{
						printf("\\n:\t");
					}
					else
					{
						printf("%c:\t",i);
					}
				}
				flag = true;
				printf("%d ", copiedarray2[i][j]);
			}
		}
		if(flag) printf("\n");
	}

	printf("+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++\n");
	getchar();
}

//Huffman table construction+++++++++++++++++++++++++++++++++++++++++++++++++++

void insertion_sort(node **forest, int length)
{	
	for(int i = 1; i < length ; i++)
	{	
		node *tmp = forest[i];
		int j = i - 1;
		bool done = false;

		do
		{
			if(forest[j]->weight < tmp->weight)		//> ascending order; < descending order
			{
				forest[j+1] = forest[j];
				j = j-1;
				if(j < 0)
				{
					done = true;
				}
			}
			else
			{
				done = true;
			}
		}while(!done);
		forest[j+1] = tmp;
	}
}

void print_char_frequency()
{
	printf("+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++\n");
	printf("character frequency:\n");
	printf("+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++\n");

	for(int i = 0 ; i < MAX_CHAR ; i++)
	{
		if(char_frequency[i] != 0)
		{
			if(i == 10)//new line
			{
				printf("%d)\tval: \\n\tfreq: %d\n",i, char_frequency[i]);
			}
			else
			{
				printf("%d)\tval: %c\tfreq: %d\n",i, i, char_frequency[i]);
			}
		}
	}

	printf("+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++\n");
}


void print_char_huffman_table()
{
	bool flag = false;

	printf("+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++\n");
	printf("Huffman table:\n");
	printf("+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++\n");
	
	for(int i = 0 ; i < MAX_CHAR ; i++)
	{		
		flag = false;
		for(int j = 0 ; j < (MAX_CHAR -1) ; j++)
		{		
			if(char_huffman_table[i][j] != -1)
			{				
				if(!flag)
				{
					if(i == 10)//new line
					{
						printf("\\n:\t");
					}
					else
					{
						printf("%c:\t",i);
					}
				}
				flag = true;
				printf("%d ", char_huffman_table[i][j]);
			}
		}
		if(flag) printf("\n");
	}

	printf("+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++\n");
}

void printout_inorder(node * tree) 
{	

	if(tree->left) printout_inorder(tree->left);

	if(tree->val != NULL)
	{
		if(tree->val == '\n')
		{
			printf("weight: %d\tvalue: \\n\n",tree->weight);
		}
		else
		{
			printf("weight: %d\tvalue: %c\n",tree->weight, tree->val);
		}
	}
	else
	{
		printf("weight: %d\tvalue: NULL\n",tree->weight);
	}

	if(tree->right) printout_inorder(tree->right);
}

void read_file()
{
	FILE *file;
	unsigned char end_of_file = 255;
	 unsigned char c;

	file = fopen(INPUT_FILE_NAME, "r");
  
	if (file==NULL)
	{
		perror ("Error reading file");		
	}
	else
	{
	//storing the file contents into h_input
		h_input = (unsigned char *)malloc(MAX_FILE_CHARS*sizeof(char));
		do
			{
			c = getc (file);
			//if(c == end_of_file) printf("\n Found EOF \n");
			//printf("c before putting into array is %c\n",c);
			h_input[number_of_char]=c;
			number_of_char++;
			char_frequency[c]++;			
		} while (c != end_of_file);

		fclose (file);		
	}
//	h_input[number_of_char] = end_of_file;
	char_frequency[end_of_file] = 0;				//to avoid problems with several EOF in one file
	//EOF is not needed ; so going to decrement
	number_of_char--;
}

void traverse_preorder(node *root, int *path)
{
	if(root->val != NULL)
	{
		for(int i = 0 ; i < MAX_CHAR -1 ; i++)
		{
			char_huffman_table[root->val][i] = path[i];
		}
	}

	if(root->left)//left 1
	{
		int counter = 0;

		for(int i = 0 ; i < MAX_CHAR - 1 ; i++)
		{			
			if(path[i] == -1)
			{
				break;
			}

			counter++;
		}

		path[counter] = 1;

		traverse_preorder(root->left, path);

		path[counter] = -1;
	}
	
	if(root->right)//right 0
	{
		int counter = 0;

		for(int i = 0 ; i < MAX_CHAR - 1 ; i++)
		{			
			if(path[i] == -1)
			{
				break;
			}

			counter++;
		}

		path[counter] = 0;

		traverse_preorder(root->right, path);

		path[counter] = -1;		
	}
}

void construct_huffman_table(node *root)
{		
	int path[MAX_CHAR - 1];
	for(int i = 0 ; i < MAX_CHAR - 1 ; i++)
	{
		path[i] = -1;
	}

	traverse_preorder(root, path);		
}


void build_binary_tree()
{
	int forest_counter = 0;
	node *forest[MAX_CHAR];
	node *curr;

	for(int i = 0 ; i < MAX_CHAR ; i++)		//initial forest
	{
		if(char_frequency[i] != 0)
		{
			curr = (node *)malloc(sizeof(node));
			curr->left = curr->right = NULL;
			curr->val = i;
			curr->weight = char_frequency[i];

			forest[forest_counter] = curr;
			forest_counter++;
		}
	}

	insertion_sort(forest, forest_counter);//sorted initial forest
		
	while(forest_counter > 1)//build final tree
	{
		node *parent;
		parent = (node *)malloc(sizeof(node));
		parent->right = forest[forest_counter-1];
		parent->left = forest[forest_counter-2];
		parent->weight = forest[forest_counter-1]->weight + forest[forest_counter-2]->weight;
		parent->val = NULL;

		forest[forest_counter-1] = NULL;
		forest[forest_counter-2] = parent;
	
		forest_counter--;

		insertion_sort(forest, forest_counter);
	}

	printf("+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++\n");
	printf("Huffman tree (inorder traversal sequence):\n");
	printf("+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++\n");
	printout_inorder(forest[0]);
	printf("+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++\n");

	construct_huffman_table(forest[0]);

	//delete_binary_tree_postorder(forest[0]);	//after building Huffman table we do not need Huffman tree anymore
}

void array_initializer()
{
	for(int i = 0 ; i < MAX_CHAR ; i++)
	{
		char_frequency[i] = 0;
	}

	for(int i = 0 ; i < MAX_CHAR ; i++)
	{
		for(int j = 0 ; j < (MAX_CHAR-1) ; j++)
		{
			char_huffman_table[i][j] = -1;
		}
	}
}


//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++

//Calculation on CPU+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++

void compress_file()
{
	FILE *input_file;
	FILE *compressed_file;
	unsigned char output_char;
	int write_counter;	//how many bits have been written in specific byte
	
	unsigned char input_char;
	unsigned char end_of_file ;
	unsigned char mask ; //00000001;
	
	

	input_file = fopen(INPUT_FILE_NAME, "rb");
	compressed_file = fopen(COMPRESSED_FILE_NAME, "wb");			// apend file (add text to a file or create a file if it does not exist)
	output_char= 0x0;
	write_counter = 0;
	end_of_file = 255;
	mask = 0x01;

  
	if ((input_file==NULL)||(compressed_file==NULL))
	{
		perror ("Error reading file");		
	}
	else
	{
		do
		{
			input_char = getc (input_file);							//read one character from input file
			
			for(int i = 0 ; i < (MAX_CHAR - 1) ; i++)				
			{
				if(char_huffman_table[input_char][i] == 0)			//detect if current character on particular position has 0 or 1
				{
					output_char = output_char << 1;					//if 0 then shift bits one position to left (last bit after shifting is 0)
					write_counter++;
				}
				else if(char_huffman_table[input_char][i] == 1)
				{
					output_char = output_char << 1;					//if 1 then shift bits one position to left...
					output_char = output_char | mask;				//...and last bit change to: 1
					write_counter++;
				}
				else //-1
				{
					if(input_char == end_of_file)					//if EOF is detected then write current result to file
					{													
						if(write_counter != 0)
						{
							output_char = output_char << (8-write_counter);
							printf("Compressed char in decimal is %d \n", output_char);
							putc(output_char, compressed_file);
							output_char = 0x0;
						}
						else	//write_counter == 0
						{
							printf("Compressed char in decimal is %d \n", output_char);
							putc(output_char, compressed_file);				
						}
					}

					break;
				}

				if(write_counter == 8)								//if result achieved 8 (size of char) then write it to compressed_file
				{
					printf("Compressed char in decimal is %d \n", output_char);
					putc(output_char, compressed_file);				
					output_char = 0x0;
					write_counter = 0;
				}
			}

		} while (input_char != end_of_file);

		fclose (input_file);
		fclose(compressed_file);

		last_byte_padding = write_counter;							//to decompress file we have to know how many bits in last byte have been written
		
		//update_config(write_counter); //TODO to zakomentowac przy ostatecznych pomiarach
	}
}


void print_gpu_compressed_file(char *final_compressed_cpu,int finalsize_cpu)
{
FILE *compressed_file;

compressed_file = fopen(COMPRESSED_FILE_NAME_GPU, "wb");	
for(int i=0;i<finalsize_cpu;i++)
{

char c=final_compressed_cpu[i];
printf("i is %d and c is %c",i,c);
putc(c,compressed_file);

}

fclose(compressed_file);

}


void decompress_file()
{
	FILE *compressed_file;
	FILE *decompressed_file;
	unsigned char end_of_file = 255;
	unsigned char mask = 0x7F; //01111111;	
	unsigned char curr;
	unsigned char next;
	int written_char_counter=0;
	int pattern[MAX_CHAR - 1];

	for(int i = 0 ; i < (MAX_CHAR - 1); i++)
	{
		pattern[i] = -1;
	}

	compressed_file = fopen(COMPRESSED_FILE_NAME, "rb");
	decompressed_file = fopen(DECOMPRESSED_FILE_NAME, "wb");

	if ((compressed_file==NULL)||(decompressed_file==NULL))
	{
		perror ("Error reading file");		
	}
	else
	{
		int bit_counter=0;		
		unsigned char first_bit;
		bool read_next = true;

		curr = getc (compressed_file);
		next = getc (compressed_file);					//we have to read one byte in advance due to padding

//		for(int i = 0 ; i < (MAX_CHAR - 1) ; i++)		//builds a pattern and chcecks if it matches to char_huffman_table
		int pattern_counter=-1;
		while(pattern_counter < (MAX_CHAR - 1))
		{ 
			pattern_counter++;

			first_bit = curr | mask;							//check if first bit is 0 or 1

			curr = curr << 1;

			if(bit_counter == 7)
			{
				bit_counter = 0;				

				curr = next;

				if(read_next)
				{
					next = getc (compressed_file);

					if(next == end_of_file)
					{
						if((number_of_char - written_char_counter) < 8)
						{
							read_next = false;
							bit_counter = 7 - last_byte_padding;					
						}
					}
				}											

				if((curr == end_of_file) && ((number_of_char - written_char_counter) < 8))
				{
					break;
				}					
			}
			else
			{
				bit_counter++;
			}

			if(first_bit == 255)
			{
				pattern[pattern_counter] = 1;
			}
			else
			{
				pattern[pattern_counter] = 0;
			}

			bool flag = true;
			for(int j = 0 ; j < MAX_CHAR ; j++)
			{
				flag = true;

				for(int k = 0 ; k < (MAX_CHAR - 1) ; k++)
				{
					if(char_huffman_table[j][k] != pattern[k])
					{
						flag = false;
						break;
					}
				}

				if(flag == true)
				{	
					written_char_counter++;					
					putc(j, decompressed_file);
					
					for(int i = 0 ; i < (MAX_CHAR - 1); i++)
					{
						pattern[i] = -1;
					}

					pattern_counter = -1;

					break;
				}
			}
				
		}

		fclose (compressed_file);		
		fclose (decompressed_file);		
	}
}


void initialize()
{	
	array_initializer();	
	read_file();	
	print_char_frequency();
	build_binary_tree();
	print_char_huffman_table();
}


/*__global__ void compress(int *d_input,int number_of_char,int *d_char_huffman_table,int MAX_CHAR)
{
	int i=0;

	 extern __shared__ int my2DArray[32][32]; //size need to be coded a development time though  
	 my2DArray[threadIdx.x][threadIdx.y] = flatArray[blockDim.x * threadIdx.y + threadIdx.x];
	
}*/


__global__ void read2darray(int *devPtr,int pitch)
{
	int elements[2][2];
	 for (int r = 0; r < 2; ++r) {
        int* row = (int*)((char*)devPtr + r * pitch);
        for (int c = 0; c < 2; ++c) {
             elements[r][c] = row[c];
        }
    }
}


__global__ void check_bool(bool *d_bool)
{
	d_bool[0]=false;
	d_bool[1]=false;
}


void print_huffman()
{

printf(" \n Huffman after copying back \n " );
unsigned char input_char1;
unsigned char input_char2;

unsigned char input_char3;
unsigned char input_char4;

input_char1 = h_input[0];
input_char2 = h_input[1];
input_char3 = h_input[2];
input_char4 = h_input[3];
for (int i=0;i < MAX_CHAR-1;i++)
{
if(huffman_check[ input_char1*255+i]!= -1 )  printf ("\t%c code is  %d  \n", input_char1,huffman_check[ input_char1*255+i]);
}


for (int i=0;i < MAX_CHAR-1;i++)
{
if(huffman_check[ input_char2*255+i]!= -1) printf ("\t%c code is  %d  \n",input_char2, huffman_check[ input_char2*255+i]);
}

for (int i=0;i < MAX_CHAR-1;i++)
{
if(huffman_check[ input_char3*255+i]!= -1 )  printf ("\t%c code is  %d  \n", input_char3,huffman_check[ input_char3*255+i]);
}

for (int i=0;i < MAX_CHAR-1;i++)
{
if(huffman_check[ input_char4*255+i]!= -1 )  printf ("\t%c code is  %d  \n", input_char4,huffman_check[ input_char4*255+i]);
}




}

int main(int argc, char* argv[])
{
	
	int *finalsize_cpu=0;
	unsigned char end_of_file = 255;
	printf("start\n");
	initialize();

	hipEvent_t start, stop;   	 	// cuda events to measure time
	float elapsed_time,elapsed_time_Cont; 

	hipEventCreate(&start);     	// timing objects
	hipEventCreate(&stop);

	unsigned int timer2=0;
	time_t seconds;


// In initialize in cpu,we put the file chars into array, fill huffman table and char_freq_arrays
//copy the input contents into an array
	hipMalloc((void **)&d_input,number_of_char*sizeof(char));
	checkCUDAError("Error in allocating d_input");	
	hipMemcpy(d_input,h_input,number_of_char*sizeof(char),hipMemcpyHostToDevice);
	checkCUDAError("Error in copying d_input");	

	// Allocate space for the compressed file to be used in GPU
	hipMalloc((void **)&compressedfile_array,number_of_char*(MAX_CHAR -1)*sizeof(bool));
	checkCUDAError("Error in allocating compressedfile_array");	
	//
	hipMalloc((void **)&d_char_huffman_table,(MAX_CHAR)*(MAX_CHAR-1) * sizeof(int));
	checkCUDAError("Error in allocating d_char_huffman_table");	
	hipMemcpy(d_char_huffman_table,char_huffman_table,(MAX_CHAR)*(MAX_CHAR-1) * sizeof(int),hipMemcpyHostToDevice);
	checkCUDAError("Error in copying d_char_huffman_table");	
	hipMemcpy(huffman_check,d_char_huffman_table,(MAX_CHAR)*(MAX_CHAR-1) * sizeof(int),hipMemcpyDeviceToHost);
	checkCUDAError("Error in copying back");	

	hipMalloc((void **)&block_cntr_array,number_of_char*sizeof(int));
	checkCUDAError("Error in allocating block_cntr_array");	

	hipMalloc((void **)&d_last_byte_padding,sizeof(int));
	checkCUDAError("Error in allocating d_last_byte_padding");	
	hipMalloc((void **)&finalsize,sizeof(int));
	checkCUDAError("Error in allocating finalsize");	
	hipMalloc((void **)&orig_number_of_char,sizeof(int));
	checkCUDAError("Error in allocating orig_number_of_char");	
	hipMemcpy(orig_number_of_char,&number_of_char,sizeof(int),hipMemcpyHostToDevice);
	checkCUDAError("Error in copying orig_number_of_char");	

	
	
	// check if i can make a boolean array
	h_bool=(bool *) malloc(2*sizeof(bool));
	h_bool[0]=true;
	h_bool[1]=true;
	printf("bool1 is %d and bool2 is %d \n",h_bool[0],h_bool[1]);
	hipMalloc((void **)&d_bool,2*sizeof(bool));
	checkCUDAError("Error in d_bool");	
	hipMemcpy(d_bool,h_bool,2*sizeof(bool),hipMemcpyHostToDevice);
	checkCUDAError("Error in copying d_bool");
	//check_bool<<<1,1>>>(d_bool);
	checkCUDAError("Error in kernel changing d_bool");
	hipDeviceSynchronize();
	checkCUDAError("Error in hipDeviceSynchronize");
	hipMemcpy(h_bool,d_bool,2*sizeof(bool),hipMemcpyDeviceToHost);
	checkCUDAError("Error in copying d_bool back");
	printf("Now bool1 is %d and bool2 is %d \n",h_bool[0],h_bool[1]);

	int checkhuff[2][3]= { {0, 0, 0},
							{1, 1, 1} };
	bool flag = true;
	for(int j=0;j<2;j++)
	{	
		flag = true;
			for(int k=0;k<2;k++)
			{
				printf("h_bool is %d \t checkhuff is %d \n",int(h_bool[k]),checkhuff[j][k]);
				if(checkhuff[j][k] != int(h_bool[k]))
				{
					flag = false;
					break;
				}
			}
			if(flag == true)
			{
				printf("pattern for %d is found\n",checkhuff[j][0]);
			}

	}
	


	//copy and send the huffman table as a 2d array to GPU Device
	//int *darray=0;
	//size_t pitch;   
	//hipMallocPitch( (void**)&darray, &pitch, 2 * sizeof(int), 2); 
	//hipMemcpy2D(darray,pitch,harray,2*sizeof(int),2*sizeof(int),2,hipMemcpyHostToDevice);
	//hipMalloc((void **)&darray,4*sizeof(int));
	//hipMemcpy(darray,harray,4*sizeof(int),hipMemcpyHostToDevice);
	//hipMemcpy2D(copiedarray,2*sizeof(int),darray,pitch,pitch,2,hipMemcpyDeviceToHost);
//	hipMemcpy2D(copiedarray,2*sizeof(int),darray,pitch,2*sizeof(int),2,hipMemcpyDeviceToHost);
	//printf("After copying back %d, \t %d, \t %d,  \t %d \n",copiedarray[0][0],copiedarray[0][1],copiedarray[1][0],copiedarray[1][1]);
	//int *darray_2d=0;
	//hipMalloc((void **)&darray_2d,4*sizeof(int));
	//hipMemcpy(darray,harray,4*sizeof(int),hipMemcpyHostToDevice);
//	read2darray<<<1,1>>>(darray, pitch); 
	
	//size_t pitch2;   
	//hipMallocPitch( (void**)&d_char_huffman_table, &pitch2, (MAX_CHAR-1) * sizeof(int), MAX_CHAR); 
	
	
	//hipMemcpy2D(d_char_huffman_table,pitch2,char_huffman_table,(MAX_CHAR-1) * sizeof(int),(MAX_CHAR-1) * sizeof(int),MAX_CHAR,hipMemcpyHostToDevice);
	
	//hipMemcpy2D(char_huffman_table_gpu,(MAX_CHAR-1) * sizeof(int),char_huffman_table,(MAX_CHAR-1) * sizeof(int),(MAX_CHAR-1) * sizeof(int),MAX_CHAR,hipMemcpyHostToDevice);
	//checkCUDAError("Error in char_huffman_table_gpu");	
	//hipMemcpy2D(copiedarray2,(MAX_CHAR-1)*sizeof(int),d_char_huffman_table,pitch2,(MAX_CHAR-1)*sizeof(int),MAX_CHAR,hipMemcpyDeviceToHost);
	
	//hipMemcpy2D(copiedarray2,(MAX_CHAR-1)*sizeof(int),char_huffman_table_gpu,(MAX_CHAR-1) * sizeof(int),(MAX_CHAR-1) * sizeof(int),MAX_CHAR,hipMemcpyDeviceToHost);
//hipMemcpy(orig_number_of_char,&number_of_char,sizeof(int),hipMemcpyHostToDevice);
//	checkCUDAError("Error in copiedarray2");	
//	print_dchar_huffman_table();
	printf("\n the number of characters in the input file is %d \n",number_of_char);
	getchar();
	/*for(int i=0;i<number_of_char;i++)
	{
		if( h_input[i] == end_of_file ) printf(" EOF \n");
		printf(" Copying into array: i is %d  and c is %c \n",i,h_input[i]);
	}*/
	


	getchar();
	print_huffman();
	
	int no_of_blocks = (number_of_char + BLOCK_SIZE -1)/BLOCK_SIZE;
		printf("no_of_blocksis %d \n", no_of_blocks);
	if(no_of_blocks == 0) no_of_blocks =1;

	//compress_file_gpu<<<number_of_char,1>>>(d_input,compressedfile_array,d_char_huffman_table,block_cntr_array,d_last_byte_padding);
	hipEventRecord(start, 0);		// start time
	checkCUDAError("Error in hipEventRecord start \n");	
	compress_file_gpu<<<no_of_blocks,BLOCK_SIZE>>>(d_input,compressedfile_array,d_char_huffman_table,block_cntr_array,d_last_byte_padding,finalsize,orig_number_of_char,number_of_char);
	checkCUDAError("Error in compress_file_gpu \n");	
	hipDeviceSynchronize();

	//hipMalloc((void **)&block_cntr_array_check,number_of_char*sizeof(int));
	//checkCUDAError("Error in allocating block_cntr_array_check");	
	block_cntr_array_check = (int *) malloc(number_of_char*sizeof(int));
	
	hipMemcpy(block_cntr_array_check,block_cntr_array,number_of_char*sizeof(int),hipMemcpyDeviceToHost);
	checkCUDAError("Error in copying back block_cntr_array_check");

	for(int i=0; i < number_of_char; i++)
	{
		printf(" block size for i = %d is %d \n",i, block_cntr_array_check[i]);
	}


//	computearray_size<<<1,1>>>(block_cntr_array,finalsize,orig_number_of_char);
	checkCUDAError("Error in Compute array \n");
		finalsize_cpu = (int *)malloc(sizeof(int));
	hipMemcpy(finalsize_cpu,finalsize,sizeof(int),hipMemcpyDeviceToHost);
	printf("The final compressed array size is %d \n ", *finalsize_cpu);
	checkCUDAError("Error in finalsize_cpu");
	int block = *finalsize_cpu;
	//allocate space for the final compressed array

	hipMalloc((void **)&finalcompressed_array,((*finalsize_cpu)*sizeof(bool)));
	checkCUDAError("hipMemcpyHostToDevice");
	
	final_compression<<<no_of_blocks,BLOCK_SIZE>>>(block_cntr_array,compressedfile_array,finalcompressed_array,number_of_char);
	checkCUDAError("Error in final_compression call \n");

	hipDeviceSynchronize();
	checkCUDAError("Error in hipDeviceSynchronize \n");
	hipEventRecord(stop, 0); 
	checkCUDAError("Error in hipEventRecord stop \n");

	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("Time to calculate results: %f ms.\n", elapsed_time);  // print out execution time

	bool *final_compressed_cpu=0;
	final_compressed_cpu = (bool *)malloc((*finalsize_cpu)*sizeof(bool));
	hipMemcpy(final_compressed_cpu,finalcompressed_array,((*finalsize_cpu)*sizeof(bool)),hipMemcpyDeviceToHost);
	checkCUDAError("Error in copying final_compressed_cpu\n");
	//print_gpu_compressed_file(final_compressed_cpu,*finalsize_cpu);
	printf("The compressed value in binary is ");

	write_GPU_compressed(final_compressed_cpu,finalsize_cpu);
	for(int i=0;i<*finalsize_cpu;i++)
	//	sprintf(compress_file+i,final_compressed_cpu[i]);

	//printf("i is %d and val is %d \n",i,final_compressed_cpu[i]);
	printf("\n");




//	hipMalloc((void **)&d_char_huffman_table,(MAX_CHAR)*(MAX_CHAR-1)*sizeof(int));
//	hipMemcpy(d_char_huffman_table,char_huffman_table,(MAX_CHAR)*(MAX_CHAR-1)*sizeof(int),hipMemcpyHostToDevice);
//	printf("\n Going to compress on the GPU ");
//	compress<<<number_of_char,1>>>(d_input,number_of_char,d_char_huffman_table,MAX_CHAR);

//test ends
	
	printf("compressing on CPU...\n");
	  //timer2=0;
	// CUT_SAFE_CALL(cutCreateTimer(&timer2));
	 //CUT_SAFE_CALL(cutStartTimer(timer2));

	/*clock_t Linuxclock_start,Linuxclock_end;		// clock return type
	hipEvent_t CUDAevent_start, CUDAevent_end;
	float CUDAEvents_time; 
	hipEventRecord(CUDAevent_start, 0 );
	hipEventSynchronize(CUDAevent_start);  
	Linuxclock_start = clock();*/
	/*time_t before,after;

	before = time (NULL);*/
clock_t start1, stop1;
	start1 = clock();
	compress_file();
	stop1 = clock();
float elapsedTime = (float)(stop1 - start1) /
(float)CLOCKS_PER_SEC * 1000.0f;
printf( "Time in cpu : %3.1f ms\n", elapsedTime );
printf("Time to calculate results: %f ms.\n", elapsed_time);  // print out execution time

printf("Speedup achieved is %lf \n", elapsedTime/elapsed_time );
	/*after = time (NULL);
	double dif;
	dif = difftime (after,before);
	printf ("It took you %.9lf seconds to type your name.\n", dif );*/



	/*Linuxclock_end = clock();
	hipEventRecord(CUDAevent_end, 0 );    	 // instrument code to measure end time
	hipEventSynchronize(CUDAevent_end);
	hipEventElapsedTime(&CUDAEvents_time, CUDAevent_start, CUDAevent_end);
	printf("CPU Time using CUDA events: %f ms\n", CUDAEvents_time);  // time_CUDAEvents is in ms
	printf("CPU Time using Linux clock: %f ms\n", ((double) (Linuxclock_end - Linuxclock_start) * 1000)/CLOCKS_PER_SEC);  //Linuxclock in sec
	hipEventDestroy(CUDAevent_start);
	hipEventDestroy(CUDAevent_end);*/

	//CUT_SAFE_CALL(cutStopTimer(timer2));
	//float time2=cutGetAverageTimerValue(timer2); 
	//printf("  Time on Host %f\n", time2);

	printf("decompressing on CPU...\n");
	decompress_file();

	getchar();
	return 0;
}
